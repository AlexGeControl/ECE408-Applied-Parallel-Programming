#include "hip/hip_runtime.h"
#include "kernel.hpp"
#include <hip/hip_runtime.h>
#include <chrono>
#include <cmath>
#include <iostream>
#include <stdexcept>

// CUDA error checking macro
#define CUDA_CHECK(call) \
    do { \
        hipError_t error = call; \
        if (error != hipSuccess) { \
            std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ << " - " << hipGetErrorString(error) << std::endl; \
            exit(1); \
        } \
    } while(0)

using Byte = unsigned char;

inline size_t getImageSize(const cv::Mat& image) {
    return static_cast<size_t>(image.rows * image.cols * sizeof(Byte));
}   

inline size_t getFilterSize(const cv::Mat& filter) {
    return static_cast<size_t>(filter.rows * filter.cols * sizeof(float));
}   

#define INPUT_TILE_SIZE 32
#define FILTER_RADIUS 1
#define OUTPUT_TILE_SIZE (INPUT_TILE_SIZE - 2 * FILTER_RADIUS)

// Declare device constant memory for filter
#define FILTER_SIZE (2 * FILTER_RADIUS + 1)

__constant__ float filterDevice[FILTER_SIZE * FILTER_SIZE];

__global__
void naiveConvolutionKernel(
    Byte* outputImage, 
    const Byte* inputImage, 
    const unsigned int height, 
    const unsigned int width
) {
    const unsigned int outputRow = blockIdx.y * blockDim.y + threadIdx.y;
    const unsigned int outputCol = blockIdx.x * blockDim.x + threadIdx.x;

    // Skip out-of-bounds pixels
    if (outputRow >= height || outputCol >= width) {
        return;
    }   

    // Do convolution
    float intensity{0.0f};
    for (int rowOffset{-FILTER_RADIUS}; rowOffset <= FILTER_RADIUS; ++rowOffset) {
        for (int colOffset{-FILTER_RADIUS}; colOffset <= FILTER_RADIUS; ++colOffset) {
            const int inputRow = static_cast<int>(outputRow) + rowOffset;
            const int inputCol = static_cast<int>(outputCol) + colOffset;

            // Skip out-of-bounds pixels
            if (inputRow < 0 || inputRow >= static_cast<int>(height) || 
                inputCol < 0 || inputCol >= static_cast<int>(width)) {
                continue;
            }

            // Compute the offset for the current pixel
            const unsigned int inputOffset = inputRow * width + inputCol;

            const int filterRow = rowOffset + FILTER_RADIUS;
            const int filterCol = colOffset + FILTER_RADIUS;
            const unsigned int filterOffset = filterRow * FILTER_SIZE + filterCol;

            intensity += static_cast<float>(inputImage[inputOffset]) * filterDevice[filterOffset];
        }
    }

    const unsigned int outputOffset = outputRow * width + outputCol;
    outputImage[outputOffset] = static_cast<Byte>(std::fmaxf(0.0f, fminf(255.0f, intensity)));
}

__global__
void tiledConvolutionKernel(
    Byte* outputImage, 
    const Byte* inputImage, 
    const int height, 
    const int width
) {
    __shared__ Byte tile[INPUT_TILE_SIZE][INPUT_TILE_SIZE];

    const int outputRow = blockIdx.y * OUTPUT_TILE_SIZE + threadIdx.y;
    const int outputCol = blockIdx.x * OUTPUT_TILE_SIZE + threadIdx.x;
    const int tileRow = threadIdx.y;
    const int tileCol = threadIdx.x;

    // Load input tile into shared memory
    {
        const int inputRow = outputRow - FILTER_RADIUS;
        const int inputCol = outputCol - FILTER_RADIUS;
        if (
            (0 <= inputRow && inputRow < height) && 
            (0 <= inputCol && inputCol < width)
        ) {
            tile[tileRow][tileCol] = inputImage[inputRow * width + inputCol];
        } else {
            tile[tileRow][tileCol] = 0;
        }
        __syncthreads();
    }
    
    // Compute convolution
    if (tileRow < FILTER_RADIUS || tileRow >= (OUTPUT_TILE_SIZE + FILTER_RADIUS) || 
        tileCol < FILTER_RADIUS || tileCol >= (OUTPUT_TILE_SIZE + FILTER_RADIUS) || 
        outputRow >= height || outputCol >= width) 
    {
        return;
    }

    float intensity{0.0f};
    for (int filterRow = 0; filterRow < FILTER_SIZE; ++filterRow) {
        for (int filterCol = 0; filterCol < FILTER_SIZE; ++filterCol) {
            const int tileRowOffset = tileRow + filterRow - FILTER_RADIUS;
            const int tileColOffset = tileCol + filterCol - FILTER_RADIUS;
            const int filterOffset = filterRow * FILTER_SIZE + filterCol;
            intensity += static_cast<float>(tile[tileRowOffset][tileColOffset]) * filterDevice[filterOffset];
        }
    }
    const int outputOffset = outputRow * width + outputCol;
    outputImage[outputOffset] = static_cast<Byte>(fmaxf(0.0f, fminf(255.0f, intensity)));
}

int64_t convolutionDevice(cv::Mat& outputImage, const cv::Mat& inputImage, const cv::Mat& filter, const bool useTiledKernel) {
    if (inputImage.empty()) {
        throw std::runtime_error("Input image is empty");
    }
    
    // Allocate memory on the device
    const int height{inputImage.rows};
    const int width{inputImage.cols};
    const size_t imageSize{getImageSize(inputImage)};

    Byte* inputImageDevice{nullptr};
    Byte* outputImageDevice{nullptr};
    
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&inputImageDevice), imageSize));
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&outputImageDevice), imageSize));

    // Copy input from host to device
    CUDA_CHECK(hipMemcpy(inputImageDevice, inputImage.data, imageSize, hipMemcpyHostToDevice));

    // Cache filter on device constant memory
    const size_t filterSize{getFilterSize(filter)};
    CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(filterDevice), filter.data, filterSize));

    auto start = std::chrono::high_resolution_clock::now();

    // Compute 2D grayscale image convolution using device kernel
    if (useTiledKernel) {
        dim3 blockDim(INPUT_TILE_SIZE, INPUT_TILE_SIZE);
        dim3 gridDim(
            (static_cast<unsigned int>(std::ceil(static_cast<float>(width) / OUTPUT_TILE_SIZE))),
            (static_cast<unsigned int>(std::ceil(static_cast<float>(height) / OUTPUT_TILE_SIZE)))
        );
        tiledConvolutionKernel<<<gridDim, blockDim>>>(outputImageDevice, inputImageDevice, height, width);
    } else
    {
        dim3 blockDim(INPUT_TILE_SIZE, INPUT_TILE_SIZE);
        dim3 gridDim(
            (static_cast<unsigned int>(std::ceil(static_cast<float>(width) / blockDim.x))),
            (static_cast<unsigned int>(std::ceil(static_cast<float>(height) / blockDim.y)))
        );
        naiveConvolutionKernel<<<gridDim, blockDim>>>(outputImageDevice, inputImageDevice, height, width);
    }
    
    // Check for kernel launch errors
    CUDA_CHECK(hipGetLastError());

    // Wait for kernel to finish
    CUDA_CHECK(hipDeviceSynchronize());

    auto end = std::chrono::high_resolution_clock::now();
    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start);
    
    // Copy output from device to host
    CUDA_CHECK(hipMemcpy(outputImage.data, outputImageDevice, imageSize, hipMemcpyDeviceToHost));

    // Free device memory
    CUDA_CHECK(hipFree(outputImageDevice));
    CUDA_CHECK(hipFree(inputImageDevice));

    return duration.count();
}
